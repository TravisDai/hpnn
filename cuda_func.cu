#include "hip/hip_runtime.h"
/* High Performance Neural Networks  -- OVHPA 2019
 * mail: hubert.valencia _at_ imass.nagoya-u.ac.jp
 * cuda_func.cu:  contains the CUDA implementation
 * of HPNN's ANN neural network routines.
*/

/*
This file is part of HPNN library.

    HPNN is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    HPNN is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <https://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "common.h"
#include "ann.h"

/*This is a CUDA area for functions*/

#define _WP  32
#define _TPW 32
#define _TPB (_TPW*_WP)
#define _KG(n) ((n+_TPB-1)/(_TPB)),_TPB

__global__ 
void dbg_print(int n, double *x){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		printf("val[%d]=%lf\n",i,x[i]);
	}
}

__global__
void sigmoid(int n, double *x){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		x[i] = 2.0/(1.0+exp(-1.0*x[i]))-1.0;
	}
#else
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		x[i] = 2.0/(1.0+exp(-1.0*x[i]))-1.0;
#endif
}
__global__
void _dsigmoid(int n, double *in, double *out){
#ifdef NO_THREADS
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < n){
                out[i] = (-0.5 * ( in[i] * in[i] - 1.0));
        }
#else
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride)
                out[i] = (-0.5 * ( in[i] * in[i] - 1.0));
#endif
}
__global__
void dsigmoid(int n, double *in, double *out){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		out[i] *= (-0.5 * ( in[i] * in[i] - 1.0));
	7}
#else
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		out[i] *= (-0.5 * ( in[i] * in[i] - 1.0));
#endif
}
__global__
void amb(int n, double *out, double *a, double *b){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		out[i] = ( a[i] - b[i] ) * ( a[i] - b[i] );
	}
#else
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride)
                out[i] = ( a[i] - b[i] ) * ( a[i] - b[i] );
#endif
}
__global__
void mul_diff(int n, double *t, double *o, double *y){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		y[i] *= ( t[i] - o[i] );
	}
#else
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride)
		y[i] *= ( t[i] - o[i] );
	
#endif
}
__global__
void zero_mv(int m,int n, double *mat,double *vec,double *res){
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	double sum=0.;
	if(tid<m){
		for(int i=0; i<n; i++) sum += vec[i]*mat[(i*m)+tid];
		res[tid]=sum;
	}
}
__global__
void zero_tmv(int m,int n, double *mat,double *vec,double *res){
        int tid=threadIdx.x+blockIdx.x*blockDim.x;
        double sum=0.;
        if(tid<m){
                for(int i=0; i<m; i++) sum += vec[i] * mat[(tid*m)+i];
                res[tid]=sum;
        }
}
/*try*/
__global__
void fw_mv_acc(int m,int n, double *mat,double *vec,double *res){
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	double sum=0.;
	if(tid<n){
		/*a full line*/
		for(int i=0; i<m; i++) sum += vec[i]*mat[(tid*m)+i];
		res[tid]=2.0/(1.0+exp(-1.0*sum))-1.0;
	}
}
__global__
void amb_acc(int n, double *out, double *a, double *b){
	extern __shared__ double sh_data[];
	int tid=threadIdx.x;
	int i=blockIdx.x*(blockDim.x*2)+threadIdx.x;
	sh_data[tid]=0.;
	if(i<n) sh_data[tid]=(a[i]-b[i])*(a[i]-b[i]) + (a[i+blockDim.x]-b[i+blockDim.x])*(a[i+blockDim.x]-b[i+blockDim.x]);
	__syncthreads();
	/*reduction in shared memory*/
	for(int s=blockDim.x/2;s>0;s>>=1){
		if(tid<s) sh_data[tid] += sh_data[tid+s];
		__syncthreads();
	}
	/*result*/
	if(tid==0) out[blockIdx.x]=sh_data[0];
}
__global__
void dsigmoid_mul_diff(int n, double *t, double *o, double *y){
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride){
		y[i] = ( t[i] - o[i] ) * (-0.5 * ( o[i] * o[i] - 1.0));
	}
	
}

__global__
void dsigmoid_mul_delta(int m,int n, double *w,double *d,double *h,double *res){
        int tid=threadIdx.x+blockIdx.x*blockDim.x;
        double sum=0.;
        if(tid<m){
                for(int i=0; i<m; i++) sum += d[i] * w[(tid*m)+i];
                res[tid]=sum * (-0.5 * ( h[tid] * h[tid] -1.0));
        }
}
/*TRY*/
__global__
void ger_acc(int m,int n, double *a,double *b,double *res){
        int tid=threadIdx.x+blockIdx.x*blockDim.x;
        if(tid<n){
		/*DGER, one line at a time*/
		for(int i=0; i<m; i++) res[(tid*m)+i]+=a[i]*b[tid];
        }
}

/*-----------------*/
/* The C interface */
/*-----------------*/
extern "C"{

#define _K (*kernel)

double cuda_array_dbg(hipblasHandle_t cublas_handle,int n,double *gpu_in){
	double res=0.1;
	hipblasDnrm2(cublas_handle,n,gpu_in,1,&res);
	return res;
}

void cuda_ann_forward_cublas(_kernel *kernel,hipblasHandle_t cublas_handle){
        int idx;
        int M;
        int N;
        double *gpu_in;
#ifdef   _CUBLAS
	double _alpha=1.0;
	double _beta =0.0;
#endif /*_CUBLAS*/
#ifdef _TIMING
hipEvent_t start, stop;
float time;
int eventflags = hipEventBlockingSync;
hipEventCreateWithFlags(&start,eventflags);
hipEventCreateWithFlags(&stop,eventflags);
hipEventRecord(start,0);
#endif
	CUDA_ALLOC(gpu_in,_K.max_index,DOUBLE);
	CUDA_G2G_CP(_K.cuda_in,gpu_in,_K.n_inputs,DOUBLE);
/*+++ I - hiddens +++*/
        for(idx=0;idx<_K.n_hiddens;idx++){
                /*GEMV + act*/
                N=_K.hiddens[idx].n_neurons;
                M=_K.hiddens[idx].n_inputs;
#ifdef   _CUBLAS
		hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.hiddens[idx].cuda_w,M,gpu_in,1,&_beta,_K.tmp_gpu,1);
		CHK_ERR(cublas_1);
		sigmoid<<<_KG(N)>>>(N,_K.tmp_gpu);
                CHK_ERR(kernel_1);
#else  /*_CUBLAS*/
		fw_mv_acc<<<_KG(N)>>>(M,N,_K.hiddens[idx].cuda_w,gpu_in,_K.tmp_gpu);
		CHK_ERR(kernel_1);
#endif /*_CUBLAS*/
		CUDA_G2G_CP(_K.tmp_gpu,gpu_in,N,DOUBLE);
        }
/*+++ II - output +++*/
        N=_K.output.n_neurons;
        M=_K.output.n_inputs;
#ifdef   _CUBLAS
	hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.output.cuda_w,M,gpu_in,1,&_beta,_K.cuda_out,1);
	CHK_ERR(cublas_2);
	sigmoid<<<_KG(N)>>>(N,_K.cuda_out);
        CHK_ERR(kernel_2);
#else  /*_CUBLAS*/
	fw_mv_acc<<<_KG(N)>>>(M,N,_K.output.cuda_w,gpu_in,_K.cuda_out);
	CHK_ERR(kernel_2);
#endif /*_CUBLAS*/
#ifdef _TIMING
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time,start,stop);
printf("cuda_ann_forward_cublas: time = %f\n",time);
#endif
//      hipDeviceSynchronize();
}
void scuda_ann_forward_cublas(_kernel *kernel,cudastreams *cudas){
	int idx,jdx;
	int M,N,red;
	int rem;
	double *gpu_in;
#ifdef _CUBLAS
        double _alpha=1.0;
        double _beta =0.0;
#endif
#ifdef _TIMING
hipEvent_t start, stop;
float time;
int eventflags = hipEventBlockingSync;
hipEventCreateWithFlags(&start,eventflags);
hipEventCreateWithFlags(&stop,eventflags);
hipEventRecord(start,0);
#endif
/*+++ I - hiddens +++*/
	CUDA_ALLOC(gpu_in,_K.max_index,DOUBLE);
	CUDA_G2G_CP(_K.cuda_in,gpu_in,_K.n_inputs,DOUBLE);
	for(idx=0;idx<_K.n_hiddens;idx++){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		red=N/cudas->cuda_n_streams;
		rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
			hipblasDgemv(cudas->cuda_handle,
				HIPBLAS_OP_T,M,red,&_alpha,_K.hiddens[idx].cuda_w+jdx*M*red,M,
				gpu_in,1,&_beta,_K.tmp_gpu+jdx*red,1);
			CHK_ERR(cublas_1);
			sigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,_K.tmp_gpu+jdx*red);
			CHK_ERR(kernel_1);
		}
		/*launch the last kernel*/
		jdx=cudas->cuda_n_streams-1;/*necessary?*/
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDgemv(cudas->cuda_handle,
			HIPBLAS_OP_T,M,red+rem,&_alpha,_K.hiddens[idx].cuda_w+jdx*M*red,M,
			gpu_in,1,&_beta,_K.tmp_gpu+jdx*red,1);
		CHK_ERR(cublas_1);
		sigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,_K.tmp_gpu+jdx*red);
		CHK_ERR(kernel_1);
		/*now wait for everyone*/
		hipDeviceSynchronize();
#else  /*_CUBLAS*/
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			fw_mv_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>
				(M,red,_K.hiddens[idx].cuda_w+jdx*M*red,gpu_in,_K.tmp_gpu+jdx*red);
			CHK_ERR(kernel_1);
		}
		jdx=cudas->cuda_n_streams-1;/*necessary?*/
		fw_mv_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>
			(M,red+rem,_K.hiddens[idx].cuda_w+jdx*M*red,gpu_in,_K.tmp_gpu+jdx*red);
		CHK_ERR(kernel_1);
		/*now wait for everyone*/
		hipDeviceSynchronize();
#endif /*_CUBLAS*/
		/*now copy back _K.tmp_gpu to gpu_in*/
		CUDA_G2G_CP(_K.tmp_gpu,gpu_in,N,DOUBLE);
		CHK_ERR(sync_1);
	}
//M=_K.output.n_inputs;
//dbg_print<<<(M+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M,gpu_in);
/*+++ II - output +++*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	red=N/cudas->cuda_n_streams;
	rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDgemv(cudas->cuda_handle,
			HIPBLAS_OP_T,M,red,&_alpha,_K.output.cuda_w+jdx*M*red,M,
			gpu_in,1,&_beta,_K.cuda_out+jdx*red,1);
		CHK_ERR(cublas_2);
		sigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,_K.cuda_out+jdx*red);
		CHK_ERR(kernel_2);
	}
	jdx=cudas->cuda_n_streams-1;/*necessary?*/
	hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
	hipblasDgemv(cudas->cuda_handle,
		HIPBLAS_OP_T,M,red+rem,&_alpha,_K.output.cuda_w+jdx*M*red,M,
		gpu_in,1,&_beta,_K.cuda_out+jdx*red,1);
	CHK_ERR(cublas_2);
	sigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,_K.cuda_out+jdx*red);
	CHK_ERR(kernel_2);
	/*now wait for everyone*/
	hipDeviceSynchronize();
#else  /*_CUBLAS*/
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		fw_mv_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>
			(M,red,_K.output.cuda_w+jdx*M*red,gpu_in,_K.cuda_out+jdx*red);
		CHK_ERR(kernel_2);
	}
	jdx=cudas->cuda_n_streams-1;/*necessary?*/
	fw_mv_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>
		(M,red+rem,_K.output.cuda_w+jdx*M*red,gpu_in,_K.cuda_out+jdx*red);
	CHK_ERR(kernel_2);
	/*now wait for everyone*/
	hipDeviceSynchronize();
#endif /*_CUBLAS*/
	CUDA_FREE(gpu_in);
#ifdef _TIMING
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time,start,stop);
printf("scuda_ann_forward_cublas: streams = %i time = %f\n",cudas->cuda_n_streams,time);
#endif
}

#define LEARN_RATE 0.01
double cuda_ann_train_cublas(_kernel *kernel,double *train,cudastreams *cudas){
	hipblasHandle_t cublas_handle=cudas->cuda_handle;
	int idx;
	int M;
	int N;
	double *tmp_gpu;
	double **hidden_vector_ptr;
	double **delta_ptr;
	double Ep =0.;
	double Epr=0.;
	/**/
	double _alpha=1.0;
	double _beta =0.0;
	/*allocate*/
	CUDA_ALLOC(tmp_gpu,_K.n_outputs,DOUBLE);
	ALLOC(hidden_vector_ptr,_K.n_hiddens,DOUBLE *);/*HOST*/
	for(idx=0;idx<_K.n_hiddens;idx++) CUDA_ALLOC(hidden_vector_ptr[idx],_K.hiddens[idx].n_neurons,DOUBLE);/*DEVICE*/
	ALLOC(delta_ptr,_K.n_hiddens+1,DOUBLE *);/*HOST*/
	for(idx=0;idx<_K.n_hiddens;idx++) CUDA_ALLOC(delta_ptr[idx],_K.hiddens[idx].n_neurons,DOUBLE);/*DEVICE*/
	CUDA_ALLOC(delta_ptr[_K.n_hiddens],_K.n_outputs,DOUBLE);/*DEVICE*/
/*+++ I - FORWARD +++*/
/*^^^ input to hidden +++*/
	N=_K.hiddens[0].n_neurons;
	M=_K.hiddens[0].n_inputs;
	hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.hiddens[0].cuda_w,M,_K.cuda_in,1,&_beta,hidden_vector_ptr[0],1);
	CHK_ERR(cublas_1);
	sigmoid<<<_KG(N)>>>(N,hidden_vector_ptr[0]);
	CHK_ERR(kernel_1);
/*^^^ hidden to hidden (if any)*/
	for(idx=1;idx<_K.n_hiddens;idx++){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		CUBLAS_ERR(hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.hiddens[idx].cuda_w,M,hidden_vector_ptr[idx-1],1,&_beta,hidden_vector_ptr[idx],1));
		CHK_ERR(cublas_2);
		sigmoid<<<_KG(N)>>>(N,hidden_vector_ptr[idx]);
		CHK_ERR(kernel_2);
	}
/*^^^ hidden to output*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	CUBLAS_ERR(hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.output.cuda_w,M,hidden_vector_ptr[_K.n_hiddens-1],1,&_beta,_K.cuda_out,1));
	CHK_ERR(cublas_3);
	sigmoid<<<_KG(N)>>>(N,_K.cuda_out);
	CHK_ERR(kernel_3);
	/*all done, calculate a preliminary error*/
	N=_K.n_outputs;
	amb<<<_KG(N)>>>(N,tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_4);
	hipblasDasum(cublas_handle,N,tmp_gpu,1,&Ep);
	CHK_ERR(cublas_4);
	//hipDeviceSynchronize();
	Ep*=0.5;
//	printf("TRAINING INITIAL ERROR: %.15f\n",Ep);
/*+++ II - DELTAS +++*/
/*^^^ output*/
	N=_K.n_outputs;
	_dsigmoid<<<_KG(N)>>>(N,_K.cuda_out,delta_ptr[_K.n_hiddens]);
	CHK_ERR(kernel_5);
	mul_diff<<<_KG(N)>>>(N,train,_K.cuda_out,delta_ptr[_K.n_hiddens]);
	CHK_ERR(kernel_6);
/*^^^ output to hidden*/
	/*! transposed (of the transposed might be transposed)*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	hipblasDgemv(cublas_handle,HIPBLAS_OP_N,M,N,&_alpha,_K.output.cuda_w,M,delta_ptr[_K.n_hiddens],1,&_beta,delta_ptr[_K.n_hiddens-1],1);
	CHK_ERR(cublas_5);
	dsigmoid<<<_KG(M)>>>(M,hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens-1]);
	CHK_ERR(kernel_7);
/*^^^ hidden to hidden (if any)*/
	if(_K.n_hiddens>1){
		for(idx=(_K.n_hiddens-2);idx>0;idx--){
			/*! transposed (of the transposed might be transposed)*/
			N=_K.hiddens[idx+1].n_neurons;
			M=_K.hiddens[idx+1].n_inputs;
			hipblasDgemv(cublas_handle,HIPBLAS_OP_N,M,N,&_alpha,_K.hiddens[idx+1].cuda_w,M,delta_ptr[idx+1],1,&_beta,delta_ptr[idx],1);
			CHK_ERR(cublas_6);
			dsigmoid<<<_KG(M)>>>(M,hidden_vector_ptr[idx],delta_ptr[idx]);
			CHK_ERR(kernel_8);
		}
		/*add zero*/
		/*! transposed (of the transposed might be transposed)*/
		N=_K.hiddens[1].n_neurons;
		M=_K.hiddens[1].n_inputs;
		hipblasDgemv(cublas_handle,HIPBLAS_OP_N,M,N,&_alpha,_K.hiddens[1].cuda_w,M,delta_ptr[1],1,&_beta,delta_ptr[0],1);
		CHK_ERR(cublas_7);
		dsigmoid<<<_KG(M)>>>(M,hidden_vector_ptr[0],delta_ptr[0]);
		CHK_ERR(kernel_9);
	}
/*+++ III - back propagation +++*/
/*^^^ output*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	_alpha=LEARN_RATE;
	hipblasDger(cublas_handle,M,N,&_alpha,hidden_vector_ptr[_K.n_hiddens-1],1,delta_ptr[_K.n_hiddens],1,_K.output.cuda_w,M);
	CHK_ERR(cublas_8);
/*^^^ hiddens*/
	for(idx=(_K.n_hiddens-1);idx>0;idx--){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		hipblasDger(cublas_handle,M,N,&_alpha,hidden_vector_ptr[idx-1],1,delta_ptr[idx],1,_K.hiddens[idx].cuda_w,M);
		CHK_ERR(cublas_9);
	}
	/*add zero*/
	N=_K.hiddens[0].n_neurons;
	M=_K.hiddens[0].n_inputs;
	hipblasDger(cublas_handle,M,N,&_alpha,_K.cuda_in,1,delta_ptr[0],1,_K.hiddens[0].cuda_w,M);
	CHK_ERR(cublas_10);
/*+++ IV - update error +++*/
	N=_K.n_outputs;
	/*>>> update cuda_out <<<*/
if(cudas->cuda_n_streams>1) scuda_ann_forward_cublas(kernel,cudas);
else cuda_ann_forward_cublas(kernel,cublas_handle);
	amb<<<_KG(N)>>>(N,tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_10);
	hipblasDasum(cublas_handle,N,tmp_gpu,1,&Epr);
	CHK_ERR(cublas_11);
	Epr*=0.5;
//	hipDeviceSynchronize();
//	fprintf(stdout,"TRAINING UPDATED ERROR: %.15f\n",Epr);
/*+++ V - cleanup +++*/
	for(idx=0;idx<_K.n_hiddens;idx++){
		CUDA_FREE(hidden_vector_ptr[idx]);
		hidden_vector_ptr[idx]=NULL;
	}
	FREE(hidden_vector_ptr);
	for(idx=0;idx<(_K.n_hiddens+1);idx++){
		CUDA_FREE(delta_ptr[idx]);
		delta_ptr[idx]=NULL;
	}
	FREE(delta_ptr);
	CUDA_FREE(tmp_gpu);
	CHK_ERR(free_1);
	return Ep-Epr;
}

double scuda_ann_train_cublas(_kernel *kernel,double *train,cudastreams *cudas){
	int idx,jdx;
	int M,N,red;
	int rem;
	double **hidden_vector_ptr;
	double **delta_ptr;
	double Ep =0.;
	double Epr=0.;
	/**/
	double _alpha=1.0;
	double _beta =0.0;
	/*allocate*/
	ALLOC(hidden_vector_ptr,_K.n_hiddens,DOUBLE *);/*HOST*/
	for(idx=0;idx<_K.n_hiddens;idx++) CUDA_ALLOC(hidden_vector_ptr[idx],_K.hiddens[idx].n_neurons,DOUBLE);/*DEVICE*/
	ALLOC(delta_ptr,_K.n_hiddens+1,DOUBLE *);/*HOST*/
	for(idx=0;idx<_K.n_hiddens;idx++) CUDA_ALLOC(delta_ptr[idx],_K.hiddens[idx].n_neurons,DOUBLE);/*DEVICE*/
	CUDA_ALLOC(delta_ptr[_K.n_hiddens],_K.n_outputs,DOUBLE);/*DEVICE*/
/*+++ I - FORWARD +++*/
/*^^^ input to hidden +++*/
	N=_K.hiddens[0].n_neurons;
	M=_K.hiddens[0].n_inputs;
	red=N/cudas->cuda_n_streams;
	rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_T,M,red,
		&_alpha,_K.hiddens[0].cuda_w+jdx*M*red,M,_K.cuda_in,1,
		&_beta,hidden_vector_ptr[0]+jdx*red,1);
		CHK_ERR(cublas_1);
		sigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,hidden_vector_ptr[0]+jdx*red);
		CHK_ERR(kernel_1);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
	hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_T,M,red+rem,
	&_alpha,_K.hiddens[0].cuda_w+jdx*M*red,M,_K.cuda_in,1,
	&_beta,hidden_vector_ptr[0]+jdx*red,1);
	CHK_ERR(cublas_1);
	sigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,hidden_vector_ptr[0]+jdx*red);
	CHK_ERR(kernel_1);
#else  /*_CUBLAS*/
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		fw_mv_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(M,red,
		_K.hiddens[0].cuda_w+jdx*M*red,_K.cuda_in,hidden_vector_ptr[0]+jdx*red);
		CHK_ERR(kernel_1);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	fw_mv_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(M,red+rem,
	_K.hiddens[0].cuda_w+jdx*M*red,_K.cuda_in,hidden_vector_ptr[0]+jdx*red);
	CHK_ERR(kernel_1);
#endif /*_CUBLAS*/
	/*now wait for everyone*/
	hipDeviceSynchronize();/*TODO: check if necessary*/
/*^^^ hidden to hidden (if any)*/
	for(idx=1;idx<_K.n_hiddens;idx++){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		red=N/cudas->cuda_n_streams;
		rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
			hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_T,M,red,
			&_alpha,_K.hiddens[idx].cuda_w+jdx*M*red,M,hidden_vector_ptr[idx-1],1,
			&_beta,hidden_vector_ptr[idx]+jdx*red,1);
			CHK_ERR(cublas_2);
			sigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,hidden_vector_ptr[idx]+jdx*red);
			CHK_ERR(kernel_2);
		}
		jdx=cudas->cuda_n_streams-1;/*useful?*/
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_T,M,red+rem,
		&_alpha,_K.hiddens[idx].cuda_w+jdx*M*red,M,hidden_vector_ptr[idx-1],1,
		&_beta,hidden_vector_ptr[idx]+jdx*red,1);
		CHK_ERR(cublas_2);
		sigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,hidden_vector_ptr[idx]+jdx*red);
		CHK_ERR(kernel_2);
#else  /*_CUBLAS*/
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			fw_mv_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(M,red,
			_K.hiddens[idx].cuda_w+jdx*M*red,hidden_vector_ptr[idx-1],hidden_vector_ptr[idx]+jdx*red);
			CHK_ERR(kernel_2);
		}
		jdx=cudas->cuda_n_streams-1;/*useful?*/
		fw_mv_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(M,red+rem,
		_K.hiddens[idx].cuda_w+jdx*M*red,hidden_vector_ptr[idx-1],hidden_vector_ptr[idx]+jdx*red);
		CHK_ERR(kernel_2);
#endif /*_CUBLAS*/
		/*now wait for everyone*/
		hipDeviceSynchronize();/*TODO: check if necessary*/
	}
/*^^^ hidden to output*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	red=N/cudas->cuda_n_streams;
	rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_T,M,red,
		&_alpha,_K.output.cuda_w+jdx*M*red,M,hidden_vector_ptr[_K.n_hiddens-1],1,
		&_beta,_K.cuda_out+jdx*red,1);
		CHK_ERR(cublas_3);
		sigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,_K.cuda_out+jdx*red);
		CHK_ERR(kernel_3);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
	hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_T,M,red+rem,
	&_alpha,_K.output.cuda_w+jdx*M*red,M,hidden_vector_ptr[_K.n_hiddens-1],1,
	&_beta,_K.cuda_out+jdx*red,1);
	CHK_ERR(cublas_3);
	sigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,_K.cuda_out+jdx*red);
#else  /*_CUBLAS*/
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		fw_mv_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(M,red,
		_K.output.cuda_w+jdx*M*red,hidden_vector_ptr[_K.n_hiddens-1],_K.cuda_out+jdx*red);
		CHK_ERR(kernel_3);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	fw_mv_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(M,red+rem,
	_K.output.cuda_w+jdx*M*red,hidden_vector_ptr[_K.n_hiddens-1],_K.cuda_out+jdx*red);
	CHK_ERR(kernel_3);
#endif /*_CUBLAS*/
	/*all done, calculate a preliminary error*/
	N=_K.n_outputs;
	/*TODO: no streams for that part?*/
#ifdef   _CUBLAS
	amb<<<_KG(N)>>>(N,_K.tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_4);
	hipblasDasum(cudas->cuda_handle,N,_K.tmp_gpu,1,&Ep);
	CHK_ERR(cublas_4);
#else  /*_CUBLAS*/
	amb_acc<<<1,_TPB/2,sizeof(double)*(_TPB)>>>(N,_K.tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_4b);
	CUDA_G2C_CP(&Ep,_K.tmp_gpu,1,double);
#endif /*_CUBLAS*/
	Ep*=0.5;
//	printf("TRAINING INITIAL ERROR: %.15f\n",Ep);
/*+++ II - DELTAS +++*/
/*^^^ output*/
	/*TODO: no streams for that part?*/
	N=_K.n_outputs;
	dsigmoid_mul_diff<<<_KG(N)>>>(N,train,_K.cuda_out,delta_ptr[_K.n_hiddens]);
	CHK_ERR(kernel_5);
/*^^^ output to hidden*/
	/*distribution over M due to transposed operations*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	red=M/cudas->cuda_n_streams;
	rem=M%cudas->cuda_n_streams;
#ifdef   _CUBLAS
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
//FIXME: WHAT?
		hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_N,red,N,
		&_alpha,_K.output.cuda_w+jdx*N*red,red /*or M?*/,delta_ptr[_K.n_hiddens],1,
		&_beta,delta_ptr[_K.n_hiddens-1]+jdx*red,1);
		CHK_ERR(cublas_5);
		dsigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens-1]+jdx*red);
		CHK_ERR(kernel_6);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
//FIXME: WHAT?
	hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_N,red+rem,N,
	&_alpha,_K.output.cuda_w+jdx*N*red,red /*or M?*/,delta_ptr[_K.n_hiddens],1,
	&_beta,delta_ptr[_K.n_hiddens-1]+jdx*red,1);
	CHK_ERR(cublas_5);
	dsigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens-1]+jdx*red);
	CHK_ERR(kernel_6);
#else  /*_CUBLAS*/
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		dsigmoid_mul_delta<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,N,_K.output.cuda_w+jdx*N*red,
		delta_ptr[_K.n_hiddens],hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens-1]+jdx*red);
		CHK_ERR(kernel_6b);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	dsigmoid_mul_delta<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,N,_K.output.cuda_w+jdx*N*red,
	delta_ptr[_K.n_hiddens],hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens-1]+jdx*red);
	CHK_ERR(kernel_6b);
#endif /*_CUBLAS*/
	/*TODO: should we sync?*/
/*^^^ hidden to hidden (if any)*/
	if(_K.n_hiddens>1){
		for(idx=(_K.n_hiddens-2);idx>0;idx--){
			N=_K.hiddens[idx+1].n_neurons;
			M=_K.hiddens[idx+1].n_inputs;
			red=M/cudas->cuda_n_streams;
			rem=M%cudas->cuda_n_streams;
#ifdef   _CUBLAS
			for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
				hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
//FIXME: WHAT?
				hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_N,red,N,
				&_alpha,_K.hiddens[idx+1].cuda_w+jdx*N*red,red /*or M?*/,delta_ptr[idx+1],1,
				&_beta,delta_ptr[idx]+jdx*red,1);
				CHK_ERR(cublas_6);
				dsigmoid<<<_KG(red)>>>(red,hidden_vector_ptr[idx],delta_ptr[idx]+jdx*red);
				CHK_ERR(kernel_7);
			}
			jdx=cudas->cuda_n_streams-1;/*useful?*/
			hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
//FIXME: WHAT?
			hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_N,red+rem,N,
			&_alpha,_K.hiddens[idx+1].cuda_w+jdx*N*red,red /*or M?*/,delta_ptr[idx+1],1,
			&_beta,delta_ptr[idx]+jdx*red,1);
			CHK_ERR(cublas_6);
			dsigmoid<<<_KG(red+rem)>>>(red+rem,hidden_vector_ptr[idx],delta_ptr[idx]+jdx*red);
			CHK_ERR(kernel_7);
#else  /*_CUBLAS*/
			for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
				dsigmoid_mul_delta<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,N,_K.hiddens[idx+1].cuda_w+jdx*N*red,
				delta_ptr[idx+1],hidden_vector_ptr[idx],delta_ptr[idx]+jdx*red);
				CHK_ERR(kernel_7b);
			}
			jdx=cudas->cuda_n_streams-1;/*useful?*/
			dsigmoid_mul_delta<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,N,_K.hiddens[idx+1].cuda_w+jdx*N*red,
			delta_ptr[idx+1],hidden_vector_ptr[idx],delta_ptr[idx]+jdx*red);
			CHK_ERR(kernel_7b);
#endif /*_CUBLAS*/
		}
		/*add zero*/
		N=_K.hiddens[1].n_neurons;
		M=_K.hiddens[1].n_inputs;
		red=M/cudas->cuda_n_streams;
		rem=M%cudas->cuda_n_streams;
#ifdef   _CUBLAS
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
//FIXME: WHAT?
			hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_N,red,N,
			&_alpha,_K.hiddens[1].cuda_w+jdx*N*red,red /*or M?*/,delta_ptr[1],1,
			&_beta,delta_ptr[0]+jdx*red,1);
			CHK_ERR(cublas_7);
			dsigmoid<<<_KG(red)>>>(red,hidden_vector_ptr[0],delta_ptr[0]+jdx*red);
			CHK_ERR(kernel_8);
		}
		jdx=cudas->cuda_n_streams-1;/*useful?*/
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
//FIXME: WHAT?
		hipblasDgemv(cudas->cuda_handle,HIPBLAS_OP_N,red+rem,N,
		&_alpha,_K.hiddens[1].cuda_w+jdx*N*red,red /*or M?*/,delta_ptr[1],1,
		&_beta,delta_ptr[0]+jdx*red,1);
		CHK_ERR(cublas_7);
		dsigmoid<<<_KG(red+rem)>>>(red+rem,hidden_vector_ptr[0],delta_ptr[0]+jdx*red);
		CHK_ERR(kernel_8);
#else  /*_CUBLAS*/
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			dsigmoid_mul_delta<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,N,_K.hiddens[1].cuda_w+jdx*N*red,
			delta_ptr[1],hidden_vector_ptr[0],delta_ptr[0]+jdx*red);
			CHK_ERR(kernel_8b);
		}
		jdx=cudas->cuda_n_streams-1;/*useful?*/
		dsigmoid_mul_delta<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,N,_K.hiddens[1].cuda_w+jdx*N*red,
		delta_ptr[1],hidden_vector_ptr[0],delta_ptr[0]+jdx*red);
		CHK_ERR(kernel_8b);
#endif /*_CUBLAS*/
	}
/*+++ III - back propagation +++*/
/*^^^ output*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	red=N/cudas->cuda_n_streams;
	rem=N%cudas->cuda_n_streams;
	_alpha=LEARN_RATE;
#ifdef   _CUBLAS
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDger(cudas->cuda_handle,M,red,&_alpha,hidden_vector_ptr[_K.n_hiddens-1],1,
		delta_ptr[_K.n_hiddens]+jdx*red,1,_K.output.cuda_w+jdx*M*red,M);
		CHK_ERR(cublas_8);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
	hipblasDger(cudas->cuda_handle,M,red+rem,&_alpha,hidden_vector_ptr[_K.n_hiddens-1],1,
	delta_ptr[_K.n_hiddens]+jdx*red,1,_K.output.cuda_w+jdx*M*red,M);
	CHK_ERR(cublas_8);
#else  /*_CUBLAS*/
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
//TODO: CHECK
		ger_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>
			(M,red,hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens]+jdx*red,_K.output.cuda_w+jdx*M*red);
		CHK_ERR(kernel_9);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	ger_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>
		(M,red+rem,hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens]+jdx*red,_K.output.cuda_w+jdx*M*red);
	CHK_ERR(kernel_9);
#endif /*_CUBLAS*/
/*^^^ hiddens*/
	for(idx=(_K.n_hiddens-1);idx>0;idx--){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		red=N/cudas->cuda_n_streams;
		rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
			hipblasDger(cudas->cuda_handle,M,red,&_alpha,hidden_vector_ptr[idx-1],1,
			delta_ptr[idx]+jdx*red,1,_K.hiddens[idx].cuda_w+jdx*M*red,M);
			CHK_ERR(cublas_9);
		}
		jdx=cudas->cuda_n_streams-1;/*useful?*/
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDger(cudas->cuda_handle,M,red+rem,&_alpha,hidden_vector_ptr[idx-1],1,
		delta_ptr[idx]+jdx*red,1,_K.hiddens[idx].cuda_w+jdx*M*red,M);
#else  /*_CUBLAS*/
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
//TODO: CHECK
			ger_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>
				(M,red,hidden_vector_ptr[idx-1],delta_ptr[idx]+jdx*red,_K.hiddens[idx].cuda_w+jdx*M*red);
			CHK_ERR(kernel_A);
		}
		jdx=cudas->cuda_n_streams-1;/*useful?*/
		ger_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>
			(M,red+rem,hidden_vector_ptr[idx-1],delta_ptr[idx]+jdx*red,_K.hiddens[idx].cuda_w+jdx*M*red);
		CHK_ERR(kernel_A);
#endif /*_CUBLAS*/
	}
	/*add zero*/
	N=_K.hiddens[0].n_neurons;
	M=_K.hiddens[0].n_inputs;
	red=N/cudas->cuda_n_streams;
	rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDger(cudas->cuda_handle,M,red,&_alpha,_K.cuda_in,1,delta_ptr[0]+jdx*red,1,_K.hiddens[0].cuda_w+jdx*M*red,M);
		CHK_ERR(cublas_9);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
	hipblasDger(cudas->cuda_handle,M,red+rem,&_alpha,_K.cuda_in,1,delta_ptr[0]+jdx*red,1,_K.hiddens[0].cuda_w+jdx*M*red,M);
	CHK_ERR(cublas_9);
#else  /*_CUBLAS*/
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
//TODO: CHECK
		ger_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>
			(M,red,_K.cuda_in,delta_ptr[0]+jdx*red,_K.hiddens[0].cuda_w+jdx*M*red);
		CHK_ERR(kernel_A);
	}
	jdx=cudas->cuda_n_streams-1;/*useful?*/
	ger_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>
		(M,red+rem,_K.cuda_in,delta_ptr[0]+jdx*red,_K.hiddens[0].cuda_w+jdx*M*red);
	CHK_ERR(kernel_A);
#endif /*_CUBLAS*/
/*+++ IV - update error +++*/
	N=_K.n_outputs;
	/*update cuda_out*/
	scuda_ann_forward_cublas(kernel,cudas);
	/*TODO: no streams for that part?*/
#ifdef   _CUBLAS
	amb<<<_KG(N)>>>(N,_K.tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_4);
	hipblasDasum(cudas->cuda_handle,N,_K.tmp_gpu,1,&Epr);
	CHK_ERR(cublas_4);
#else  /*_CUBLAS*/
	amb_acc<<<1,_TPB/2,sizeof(double)*(_TPB)>>>(N,_K.tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_4b);
	CUDA_G2C_CP(&Epr,_K.tmp_gpu,1,double);
#endif /*_CUBLAS*/
	Epr*=0.5;
//	fprintf(stdout,"TRAINING UPDATED ERROR: %.15f\n",Epr);
/*+++ V - cleanup +++*/
	return Ep-Epr;
}




void cuda_ann_act(double *out,int size){
	sigmoid<<<(size+255)/256, 256>>>(size, out);
	CHK_ERR(sigmoid);
}
void cuda_ann_dact(double *in,double *out,int size){
	dsigmoid<<<(size+255)/256, 256>>>(size, in, out);
	CHK_ERR(dsigmoid);
}
void cuda_ann_amb(double *out, double *a,double *b,int size){
	amb<<<(size+255)/256, 256>>>(size, out, a, b);
	CHK_ERR(amb);
}
void cuda_ann_mul_diff(double *train, double *out, double *res, int size){
	mul_diff<<<(size+255)/256, 256>>>(size,train,out,res);
	CHK_ERR(mul_diff);
}
void cuda_zero_mv(int m,int n,double *mat,double *vec, double *res){
	zero_mv<<<m/256+1, 256>>>(m,n,mat,vec,res);
	CHK_ERR(zero_mv);
}

void cuda_zero_tmv(int m,int n,double *mat,double *vec, double *res){
        zero_tmv<<<n/256+1, 256>>>(m,n,mat,vec,res);
	CHK_ERR(zero_tmv);
}






}/*extern "C"*/
