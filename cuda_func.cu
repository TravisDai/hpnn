#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "common.h"
#include "ann.h"

/*This is a CUDA area for functions*/

#ifdef _CUBLAS
#warning "CUBLAS uses the Painful column-order format: GPU and CPU kernel memory will differ!"
#endif

#define _WP  32
#define _TPW 32
#define _TPB (_TPW*_WP)
#define _KG(n) ((n+_TPB-1)/(_TPB)),_TPB

__global__ 
void dbg_print(int n, double *x){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		printf("val[%d]=%lf\n",i,x[i]);
	}
}

__global__
void sigmoid(int n, double *x){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		x[i] = 2.0/(1.0+exp(-1.0*x[i]))-1.0;
	}
#else
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		x[i] = 2.0/(1.0+exp(-1.0*x[i]))-1.0;
#endif
}
__global__
void _dsigmoid(int n, double *in, double *out){
#ifdef NO_THREADS
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < n){
                out[i] = (-0.5 * ( in[i] * in[i] - 1.0));
        }
#else
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride)
                out[i] = (-0.5 * ( in[i] * in[i] - 1.0));
#endif
}
__global__
void dsigmoid(int n, double *in, double *out){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		out[i] *= (-0.5 * ( in[i] * in[i] - 1.0));
	7}
#else
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		out[i] *= (-0.5 * ( in[i] * in[i] - 1.0));
#endif
}
__global__
void amb(int n, double *out, double *a, double *b){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		out[i] = ( a[i] - b[i] ) * ( a[i] - b[i] );
//		printf("[%d, %d]:\ta=%lf\tb=%lf\tout=%lf\n",\
            blockIdx.y*gridDim.x+blockIdx.x,\
            threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
            a[i],b[i],out[i]);
	}
#else
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride)
                out[i] = ( a[i] - b[i] ) * ( a[i] - b[i] );
#endif
}
__global__
void mul_diff(int n, double *t, double *o, double *y){
#ifdef NO_THREADS
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		y[i] *= ( t[i] - o[i] );
	}
#else
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride)
		y[i] *= ( t[i] - o[i] );
	
#endif
}
__global__
void zero_mv(int m,int n, double *mat,double *vec,double *res){
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	double sum=0.;
	if(tid<m){
		for(int i=0; i<n; i++) sum += vec[i]*mat[(i*m)+tid];
		res[tid]=sum;
	}
}
__global__
void zero_tmv(int m,int n, double *mat,double *vec,double *res){
        int tid=threadIdx.x+blockIdx.x*blockDim.x;
        double sum=0.;
        if(tid<m){
                for(int i=0; i<m; i++) sum += vec[i] * mat[(tid*m)+i];
                res[tid]=sum;
        }
}
/*try*/
__global__
void fw_mv_acc(int m,int n, double *mat,double *vec,double *res){
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	double sum=0.;
	if(tid<n){
		/*a full line*/
		for(int i=0; i<m; i++) sum += vec[i]*mat[(tid*m)+i];
		res[tid]=2.0/(1.0+exp(-1.0*sum))-1.0;
	}
}




/*-----------------*/
/* The C interface */
/*-----------------*/
extern "C"{

#define _K (*kernel)

double cuda_array_dbg(hipblasHandle_t cublas_handle,int n,double *gpu_in){
	double res=0.1;
	hipblasDnrm2(cublas_handle,n,gpu_in,1,&res);
	return res;
}

void cuda_ann_forward_cublas(_kernel *kernel,hipblasHandle_t cublas_handle){
        int idx;
        int M;
        int N;
        double *gpu_in;
#ifdef   _CUBLAS
	double _alpha=1.0;
	double _beta =0.0;
#endif /*_CUBLAS*/
#ifdef _TIMING
hipEvent_t start, stop;
float time;
int eventflags = hipEventBlockingSync;
hipEventCreateWithFlags(&start,eventflags);
hipEventCreateWithFlags(&stop,eventflags);
hipEventRecord(start,0);
#endif
	CUDA_ALLOC(gpu_in,_K.max_index,DOUBLE);
	CUDA_G2G_CP(_K.cuda_in,gpu_in,_K.n_inputs,DOUBLE);
/*+++ I - hiddens +++*/
        for(idx=0;idx<_K.n_hiddens;idx++){
                /*GEMV + act*/
                N=_K.hiddens[idx].n_neurons;
                M=_K.hiddens[idx].n_inputs;
#ifdef   _CUBLAS
		hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.hiddens[idx].cuda_w,M,gpu_in,1,&_beta,_K.tmp_gpu,1);
		CHK_ERR(cublas_1);
		sigmoid<<<_KG(N)>>>(N,_K.tmp_gpu);
                CHK_ERR(kernel_1);
#else  /*_CUBLAS*/
		fw_mv_acc<<<_KG(N)>>>(M,N,_K.hiddens[idx].cuda_w,gpu_in,_K.tmp_gpu);
		CHK_ERR(kernel_1);
#endif /*_CUBLAS*/
		CUDA_G2G_CP(_K.tmp_gpu,gpu_in,N,DOUBLE);
        }
/*+++ II - output +++*/
        N=_K.output.n_neurons;
        M=_K.output.n_inputs;
#ifdef   _CUBLAS
	hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.output.cuda_w,M,gpu_in,1,&_beta,_K.cuda_out,1);
	CHK_ERR(cublas_2);
	sigmoid<<<_KG(N)>>>(N,_K.cuda_out);
        CHK_ERR(kernel_2);
#else  /*_CUBLAS*/
	fw_mv_acc<<<_KG(N)>>>(M,N,_K.output.cuda_w,gpu_in,_K.cuda_out);
	CHK_ERR(kernel_2);
#endif /*_CUBLAS*/
#ifdef _TIMING
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time,start,stop);
printf("cuda_ann_forward_cublas: time = %f\n",time);
#endif
//      hipDeviceSynchronize();
}
void scuda_ann_forward_cublas(_kernel *kernel,cudastreams *cudas){
	int idx,jdx;
	int M,N,red;
	int rem;
	double *gpu_in;
#ifdef _CUBLAS
        double _alpha=1.0;
        double _beta =0.0;
#endif
#ifdef _TIMING
hipEvent_t start, stop;
float time;
int eventflags = hipEventBlockingSync;
hipEventCreateWithFlags(&start,eventflags);
hipEventCreateWithFlags(&stop,eventflags);
hipEventRecord(start,0);
#endif
/*+++ I - hiddens +++*/
	CUDA_ALLOC(gpu_in,_K.max_index,DOUBLE);
	CUDA_G2G_CP(_K.cuda_in,gpu_in,_K.n_inputs,DOUBLE);
	for(idx=0;idx<_K.n_hiddens;idx++){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		red=N/cudas->cuda_n_streams;
		rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
			hipblasDgemv(cudas->cuda_handle,
				HIPBLAS_OP_T,M,red,&_alpha,_K.hiddens[idx].cuda_w+jdx*M*red,M,
				gpu_in,1,&_beta,_K.tmp_gpu+jdx*red,1);
			CHK_ERR(cublas_1);
			sigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,_K.tmp_gpu+jdx*red);
			CHK_ERR(kernel_1);
		}
		/*launch the last kernel*/
		jdx=cudas->cuda_n_streams-1;/*necessary?*/
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDgemv(cudas->cuda_handle,
			HIPBLAS_OP_T,M,red+rem,&_alpha,_K.hiddens[idx].cuda_w+jdx*M*red,M,
			gpu_in,1,&_beta,_K.tmp_gpu+jdx*red,1);
		CHK_ERR(cublas_1);
		sigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,_K.tmp_gpu+jdx*red);
		CHK_ERR(kernel_1);
		/*now wait for everyone*/
		hipDeviceSynchronize();
#else  /*_CUBLAS*/
		for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
			fw_mv_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>
				(M,red,_K.hiddens[idx].cuda_w+jdx*M*red,gpu_in,_K.tmp_gpu+jdx*red);
			CHK_ERR(kernel_1);
		}
		jdx=cudas->cuda_n_streams-1;/*necessary?*/
		fw_mv_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>
			(M,red+rem,_K.hiddens[idx].cuda_w+jdx*M*red,gpu_in,_K.tmp_gpu+jdx*red);
		CHK_ERR(kernel_1);
		/*now wait for everyone*/
		hipDeviceSynchronize();
#endif /*_CUBLAS*/
		/*now copy back _K.tmp_gpu to gpu_in*/
		CUDA_G2G_CP(_K.tmp_gpu,gpu_in,N,DOUBLE);
		CHK_ERR(sync_1);
	}
//M=_K.output.n_inputs;
//dbg_print<<<(M+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M,gpu_in);
/*+++ II - output +++*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	red=N/cudas->cuda_n_streams;
	rem=N%cudas->cuda_n_streams;
#ifdef   _CUBLAS
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
		hipblasDgemv(cudas->cuda_handle,
			HIPBLAS_OP_T,M,red,&_alpha,_K.output.cuda_w+jdx*M*red,M,
			gpu_in,1,&_beta,_K.cuda_out+jdx*red,1);
		CHK_ERR(cublas_2);
		sigmoid<<<_KG(red),0,cudas->cuda_streams[jdx]>>>(red,_K.cuda_out+jdx*red);
		CHK_ERR(kernel_2);
	}
	jdx=cudas->cuda_n_streams-1;/*necessary?*/
	hipblasSetStream(cudas->cuda_handle,cudas->cuda_streams[jdx]);
	hipblasDgemv(cudas->cuda_handle,
		HIPBLAS_OP_T,M,red+rem,&_alpha,_K.output.cuda_w+jdx*M*red,M,
		gpu_in,1,&_beta,_K.cuda_out+jdx*red,1);
	CHK_ERR(cublas_2);
	sigmoid<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>(red+rem,_K.cuda_out+jdx*red);
	CHK_ERR(kernel_2);
	/*now wait for everyone*/
	hipDeviceSynchronize();
#else  /*_CUBLAS*/
	for(jdx=0;jdx<cudas->cuda_n_streams-1;jdx++){
		fw_mv_acc<<<_KG(red),0,cudas->cuda_streams[jdx]>>>
			(M,red,_K.output.cuda_w+jdx*M*red,gpu_in,_K.cuda_out+jdx*red);
		CHK_ERR(kernel_2);
	}
	jdx=cudas->cuda_n_streams-1;/*necessary?*/
	fw_mv_acc<<<_KG(red+rem),0,cudas->cuda_streams[jdx]>>>
		(M,red+rem,_K.output.cuda_w+jdx*M*red,gpu_in,_K.cuda_out+jdx*red);
	CHK_ERR(kernel_2);
	/*now wait for everyone*/
	hipDeviceSynchronize();
#endif /*_CUBLAS*/
	CUDA_FREE(gpu_in);
#ifdef _TIMING
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time,start,stop);
printf("scuda_ann_forward_cublas: streams = %i time = %f\n",cudas->cuda_n_streams,time);
#endif
}

#define LEARN_RATE 0.01
double cuda_ann_train_cublas(_kernel *kernel,double *train,cudastreams *cudas){
	hipblasHandle_t cublas_handle=cudas->cuda_handle;
	int idx;
	int M;
	int N;
	double *tmp_gpu;
	double **hidden_vector_ptr;
	double **delta_ptr;
	double Ep =0.;
	double Epr=0.;
	/**/
	double _alpha=1.0;
	double _beta =0.0;
	/*allocate*/
	CUDA_ALLOC(tmp_gpu,_K.n_outputs,DOUBLE);
	ALLOC(hidden_vector_ptr,_K.n_hiddens,DOUBLE *);/*HOST*/
	for(idx=0;idx<_K.n_hiddens;idx++) CUDA_ALLOC(hidden_vector_ptr[idx],_K.hiddens[idx].n_neurons,DOUBLE);/*DEVICE*/
	ALLOC(delta_ptr,_K.n_hiddens+1,DOUBLE *);/*HOST*/
	for(idx=0;idx<_K.n_hiddens;idx++) CUDA_ALLOC(delta_ptr[idx],_K.hiddens[idx].n_neurons,DOUBLE);/*DEVICE*/
	CUDA_ALLOC(delta_ptr[_K.n_hiddens],_K.n_outputs,DOUBLE);/*DEVICE*/
/*+++ I - FORWARD +++*/
/*^^^ input to hidden +++*/
	N=_K.hiddens[0].n_neurons;
	M=_K.hiddens[0].n_inputs;
	hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.hiddens[0].cuda_w,M,_K.cuda_in,1,&_beta,hidden_vector_ptr[0],1);
	CHK_ERR(cublas_1);
	sigmoid<<<_KG(N)>>>(N,hidden_vector_ptr[0]);
	CHK_ERR(kernel_1);
/*^^^ hidden to hidden (if any)*/
	for(idx=1;idx<_K.n_hiddens;idx++){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		CUBLAS_ERR(hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.hiddens[idx].cuda_w,M,hidden_vector_ptr[idx-1],1,&_beta,hidden_vector_ptr[idx],1));
		CHK_ERR(cublas_2);
		sigmoid<<<_KG(N)>>>(N,hidden_vector_ptr[idx]);
		CHK_ERR(kernel_2);
	}
/*^^^ hidden to output*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	CUBLAS_ERR(hipblasDgemv(cublas_handle,HIPBLAS_OP_T,M,N,&_alpha,_K.output.cuda_w,M,hidden_vector_ptr[_K.n_hiddens-1],1,&_beta,_K.cuda_out,1));
	CHK_ERR(cublas_3);
	sigmoid<<<_KG(N)>>>(N,_K.cuda_out);
	CHK_ERR(kernel_3);
	/*all done, calculate a preliminary error*/
	N=_K.n_outputs;
	amb<<<_KG(N)>>>(N,tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_4);
	hipblasDasum(cublas_handle,N,tmp_gpu,1,&Ep);
	CHK_ERR(cublas_4);
	//hipDeviceSynchronize();
	Ep*=0.5;
//	printf("TRAINING INITIAL ERROR: %.15f\n",Ep);
/*+++ II - DELTAS +++*/
/*^^^ output*/
	N=_K.n_outputs;
	_dsigmoid<<<_KG(N)>>>(N,_K.cuda_out,delta_ptr[_K.n_hiddens]);
	CHK_ERR(kernel_5);
	mul_diff<<<_KG(N)>>>(N,train,_K.cuda_out,delta_ptr[_K.n_hiddens]);
	CHK_ERR(kernel_6);
/*^^^ output to hidden*/
	/*! transposed (of the transposed might be transposed)*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	hipblasDgemv(cublas_handle,HIPBLAS_OP_N,M,N,&_alpha,_K.output.cuda_w,M,delta_ptr[_K.n_hiddens],1,&_beta,delta_ptr[_K.n_hiddens-1],1);
	CHK_ERR(cublas_5);
	dsigmoid<<<_KG(M)>>>(M,hidden_vector_ptr[_K.n_hiddens-1],delta_ptr[_K.n_hiddens-1]);
	CHK_ERR(kernel_7);
/*^^^ hidden to hidden (if any)*/
	if(_K.n_hiddens>1){
		for(idx=(_K.n_hiddens-2);idx>0;idx--){
			/*! transposed (of the transposed might be transposed)*/
			N=_K.hiddens[idx+1].n_neurons;
			M=_K.hiddens[idx+1].n_inputs;
			hipblasDgemv(cublas_handle,HIPBLAS_OP_N,M,N,&_alpha,_K.hiddens[idx+1].cuda_w,M,delta_ptr[idx+1],1,&_beta,delta_ptr[idx],1);
			CHK_ERR(cublas_6);
			dsigmoid<<<_KG(M)>>>(M,hidden_vector_ptr[idx],delta_ptr[idx]);
			CHK_ERR(kernel_8);
		}
		/*add zero*/
		/*! transposed (of the transposed might be transposed)*/
		N=_K.hiddens[1].n_neurons;
		M=_K.hiddens[1].n_inputs;
		hipblasDgemv(cublas_handle,HIPBLAS_OP_N,M,N,&_alpha,_K.hiddens[1].cuda_w,M,delta_ptr[1],1,&_beta,delta_ptr[0],1);
		CHK_ERR(cublas_7);
		dsigmoid<<<_KG(M)>>>(M,hidden_vector_ptr[0],delta_ptr[0]);
		CHK_ERR(kernel_9);
	}
/*+++ III - back propagation +++*/
/*^^^ output*/
	N=_K.output.n_neurons;
	M=_K.output.n_inputs;
	_alpha=LEARN_RATE;
	hipblasDger(cublas_handle,M,N,&_alpha,hidden_vector_ptr[_K.n_hiddens-1],1,delta_ptr[_K.n_hiddens],1,_K.output.cuda_w,M);
	CHK_ERR(cublas_8);
/*^^^ hiddens*/
	for(idx=(_K.n_hiddens-1);idx>0;idx--){
		N=_K.hiddens[idx].n_neurons;
		M=_K.hiddens[idx].n_inputs;
		hipblasDger(cublas_handle,M,N,&_alpha,hidden_vector_ptr[idx-1],1,delta_ptr[idx],1,_K.hiddens[idx].cuda_w,M);
		CHK_ERR(cublas_9);
	}
	/*add zero*/
	N=_K.hiddens[0].n_neurons;
	M=_K.hiddens[0].n_inputs;
	hipblasDger(cublas_handle,M,N,&_alpha,_K.cuda_in,1,delta_ptr[0],1,_K.hiddens[0].cuda_w,M);
	CHK_ERR(cublas_10);
/*+++ IV - update error +++*/
	N=_K.n_outputs;
	/*>>> update cuda_out <<<*/
if(cudas->cuda_n_streams>1) scuda_ann_forward_cublas(kernel,cudas);
else cuda_ann_forward_cublas(kernel,cublas_handle);
	amb<<<_KG(N)>>>(N,tmp_gpu,train,_K.cuda_out);
	CHK_ERR(kernel_10);
	hipblasDasum(cublas_handle,N,tmp_gpu,1,&Epr);
	CHK_ERR(cublas_11);
	Epr*=0.5;
//	hipDeviceSynchronize();
//	fprintf(stdout,"TRAINING UPDATED ERROR: %.15f\n",Epr);
/*+++ V - cleanup +++*/
	for(idx=0;idx<_K.n_hiddens;idx++){
		CUDA_FREE(hidden_vector_ptr[idx]);
		hidden_vector_ptr[idx]=NULL;
	}
	FREE(hidden_vector_ptr);
	for(idx=0;idx<(_K.n_hiddens+1);idx++){
		CUDA_FREE(delta_ptr[idx]);
		delta_ptr[idx]=NULL;
	}
	FREE(delta_ptr);
	CUDA_FREE(tmp_gpu);
	CHK_ERR(free_1);
	return Ep-Epr;
}



void cuda_ann_act(double *out,int size){
	sigmoid<<<(size+255)/256, 256>>>(size, out);
	CHK_ERR(sigmoid);
}
void cuda_ann_dact(double *in,double *out,int size){
	dsigmoid<<<(size+255)/256, 256>>>(size, in, out);
	CHK_ERR(dsigmoid);
}
void cuda_ann_amb(double *out, double *a,double *b,int size){
	amb<<<(size+255)/256, 256>>>(size, out, a, b);
	CHK_ERR(amb);
}
void cuda_ann_mul_diff(double *train, double *out, double *res, int size){
	mul_diff<<<(size+255)/256, 256>>>(size,train,out,res);
	CHK_ERR(mul_diff);
}
void cuda_zero_mv(int m,int n,double *mat,double *vec, double *res){
	zero_mv<<<m/256+1, 256>>>(m,n,mat,vec,res);
	CHK_ERR(zero_mv);
}

void cuda_zero_tmv(int m,int n,double *mat,double *vec, double *res){
        zero_tmv<<<n/256+1, 256>>>(m,n,mat,vec,res);
	CHK_ERR(zero_tmv);
}






}/*extern "C"*/
